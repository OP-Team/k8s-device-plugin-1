
#include <hip/hip_runtime.h>
__global__ void add(int *deviceCount);

extern "C" { int DeviceGetCount(void) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    return deviceCount;
};}
